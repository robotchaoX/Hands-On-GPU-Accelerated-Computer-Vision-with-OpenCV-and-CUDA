#include "hip/hip_runtime.h"
#include "stdio.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 1024
#define threadsPerBlock 512

__global__ void gpu_dot(float *d_a, float *d_b, float *d_c) {
	//Declare shared memory
	__shared__ float partial_sum[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//Calculate index for shared memory 
	int index = threadIdx.x;
	//Calculate Partial Sum
	float sum = 0;
	while (tid < N) 
	{
		sum += d_a[tid] * d_b[tid];
		tid += blockDim.x * gridDim.x;
	}
	// Store partial sum in shared memory
	partial_sum[index] = sum; // 每个线程的部分和结果
	// synchronize threads 
	__syncthreads();
	// Calculating partial sum for whole block in reduce operation
	int i = blockDim.x / 2;
	while (i != 0) {
		if (index < i)
			partial_sum[index] += partial_sum[index + i]; // 循环相邻奇偶相加
		__syncthreads();
		i /= 2;
	}
	//Store block partial sum in global memory
	if (index == 0)
		d_c[blockIdx.x] = partial_sum[0]; // 当前块的总体部分和结果
}

int main(void) {
	//Declare Host Array
	float *h_a, *h_b, h_c, *partial_sum;
	//Declare device Array
	float *d_a, *d_b, *d_partial_sum;
	//Calculate total number of blocks per grid
	int block_calc = (N + threadsPerBlock - 1) / threadsPerBlock; // 除法向上取整
	int blocksPerGrid = (32 < block_calc ? 32 : block_calc); // 取块数量限制32？？
	// allocate memory on the host side
	h_a = (float*)malloc(N * sizeof(float));
	h_b = (float*)malloc(N * sizeof(float));
	partial_sum = (float*)malloc(blocksPerGrid * sizeof(float));
	// allocate the memory on the device
	hipMalloc((void**)&d_a, N * sizeof(float));
	hipMalloc((void**)&d_b, N * sizeof(float));
	hipMalloc((void**)&d_partial_sum, blocksPerGrid * sizeof(float));
	// fill the host array with data
	for (int i = 0; i<N; i++) {
		h_a[i] = i;
		h_b[i] = 2;
	}
	//Copy Array from host to device
	hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
	//Call kernel 
	gpu_dot << <blocksPerGrid, threadsPerBlock >> >(d_a, d_b, d_partial_sum);
	// copy the array back to host memory
	hipMemcpy(partial_sum, d_partial_sum, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);
	//Calculate final dot product on host
	h_c = 0;
	for (int i = 0; i<blocksPerGrid; i++) {
		h_c += partial_sum[i];
	}
	printf("The computed dot product is: %f\n", h_c);
#define cpu_sum(x) (x*(x+1))
	if (h_c == cpu_sum((float)(N - 1)))
	{
		printf("The dot product computed by GPU is correct\n");
	}
	else
	{
		printf("Error in dot product computation");
	}
	// free memory on host and device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_partial_sum);
	free(h_a);
	free(h_b);
	free(partial_sum);
}