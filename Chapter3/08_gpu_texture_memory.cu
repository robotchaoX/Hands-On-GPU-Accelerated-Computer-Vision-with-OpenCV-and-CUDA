#include "hip/hip_runtime.h"
#include "stdio.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 10
#define N 10
texture <float, 1, hipReadModeElementType> textureRef;
__global__ void gpu_texture_memory(int n, float *d_out)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < n) {
		float temp = tex1D(textureRef, float(idx));
		d_out[idx] = temp;
	}
}

int main()
{
	//Calculate number of blocks to launch
	int num_blocks = N / NUM_THREADS + ((N % NUM_THREADS) ? 1 : 0); // 除法向上取整
	//Declare device pointer
	float *d_out;
	// allocate space on the device for the result
	hipMalloc((void**)&d_out, sizeof(float) * N);
	// allocate space on the host for the results
	float *h_out = (float*)malloc(sizeof(float)*N);
	//Declare and initialize host array
	float h_in[N];
	for (int i = 0; i < N; i++) {
		h_in[i] = float(i);
	}
	//Define CUDA Array
	hipArray *cu_Array;
	hipMallocArray(&cu_Array, &textureRef.channelDesc, N, 1);
	//Copy data to CUDA Array
	hipMemcpyToArray(cu_Array, 0, 0, h_in, sizeof(float)*N, hipMemcpyHostToDevice);
	// bind a texture to the CUDA array
	hipBindTextureToArray(textureRef, cu_Array);
	//Call Kernel	
  	gpu_texture_memory << <num_blocks, NUM_THREADS >> >(N, d_out);
	// copy result back to host
	hipMemcpy(h_out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);
	printf("Use of Texture memory on GPU: \n");
	for (int i = 0; i < N; i++) {
		printf("Texture element at %d is : %f\n",i, h_out[i]);
	}
	free(h_out);
	hipFree(d_out);
	hipFreeArray(cu_Array);
	hipUnbindTexture(textureRef);
	return 0;
}
