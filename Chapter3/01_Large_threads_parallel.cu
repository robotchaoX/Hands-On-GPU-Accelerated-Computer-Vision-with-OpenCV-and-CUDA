#include "hip/hip_runtime.h"
#include "stdio.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//Defining number of elements in Array
#define N	50000 // 任意大

//Defining Kernel function for vector addition
__global__ void gpuAdd(int *d_a, int *d_b, int *d_c) {
	//Getting block index of current kernel
	int tid = threadIdx.x + blockIdx.x * blockDim.x;	
	while (tid < N) // 固定线程数量，N大小未知
	{
		d_c[tid] = d_a[tid] + d_b[tid];
		tid += blockDim.x * gridDim.x; // 递增启动的线程数量
	}
}

int main(void) {
	//Defining host arrays
	int h_a[N], h_b[N], h_c[N];
	//Defining device pointers
	int *d_a, *d_b, *d_c;
	// allocate the memory
	hipMalloc((void**)&d_a, N * sizeof(int));
	hipMalloc((void**)&d_b, N * sizeof(int));
	hipMalloc((void**)&d_c, N * sizeof(int));
	//Initializing Arrays
	for (int i = 0; i < N; i++) {
		h_a[i] = 2 * i*i;
		h_b[i] = i;
	}
	// Copy input arrays from host to device memory
	hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);
	//Calling kernels with N blocks and one thread per block, passing device pointers as parameters
	gpuAdd << <512, 512 >> >(d_a, d_b, d_c); 
	//Copy result back to host memory from device memory
	hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	int Correct = 1;
	printf("Vector addition on GPU \n");
	//Printing result on console
	for (int i = 0; i < N; i++) {
		if ((h_a[i] + h_b[i] != h_c[i]))
		{
			Correct = 0;
		}
	}
	if (Correct == 1)
	{
		printf("GPU has computed Sum Correctly\n");
	}
	else
	{
		printf("There is an Error in GPU Computation\n");
	}
	//Free up memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
