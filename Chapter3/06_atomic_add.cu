#include "hip/hip_runtime.h"
#include <stdio.h>

#define NUM_THREADS 10000
#define SIZE  10
#define BLOCK_WIDTH 100

__global__ void gpu_increment_atomic(int *d_a)
{
	// Calculate thread id for current thread
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// each thread increments elements wrapping at SIZE variable
	tid = tid % SIZE;
	// 多线程同时写同一块内存，原子加法操作
	atomicAdd(&d_a[tid], 1);
}

int main(int argc, char **argv)
{
	printf("%d total threads in %d blocks writing into %d array elements\n",
		NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, SIZE);
	// declare and allocate host memory
	int h_a[SIZE];
	const int ARRAY_BYTES = SIZE * sizeof(int);
	// declare and allocate GPU memory
	int * d_a;
	hipMalloc((void **)&d_a, ARRAY_BYTES);
	//Initialize GPU memory to zero
	hipMemset((void *)d_a, 0, ARRAY_BYTES);
	// launch the kernel
	gpu_increment_atomic << <NUM_THREADS / BLOCK_WIDTH, BLOCK_WIDTH >> >(d_a);
	// copy back the array to host memory
	hipMemcpy(h_a, d_a, ARRAY_BYTES, hipMemcpyDeviceToHost);	
	printf("Number of times a particular Array index has been incremented is: \n");
	for (int i = 0; i < SIZE; i++) 
	{ 
		printf("index: %d --> %d times\n ", i, h_a[i]); 
	}
	hipFree(d_a);
	return 0;
}
