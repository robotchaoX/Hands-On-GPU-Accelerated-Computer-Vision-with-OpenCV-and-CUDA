#include "hip/hip_runtime.h"
#include <stdio.h>

// Defining number of elements in Array
#define N 5

__global__ void gpu_local_memory(int d_in)
{
	// Define local memory
	int t_local;
	t_local = d_in * threadIdx.x;
	printf("Value of Local variable in current thread is: %d \n", t_local);
}

int main(int argc, char **argv)
{
	printf("Use of Local Memory on GPU:\n");
	// launch the kernel
	gpu_local_memory << <1, N >> >(5);  
	// waiting for all kernels to finish
	hipDeviceSynchronize();
	return 0;
}
