#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory>
#include <iostream>
#include <hip/hip_runtime.h>

int main(int argc, char **argv)
{
	printf(" CUDA Device Query (Runtime API) version (CUDART static linking)\n\n");
	int device_Count = 0;
	hipGetDeviceCount(&device_Count);
	// This function call returns 0 if there are no CUDA capable devices.
	if (device_Count == 0)
	{
		printf("There are no available device(s) that support CUDA\n");
	}
	else
	{
		printf("Detected %d CUDA Capable device(s)\n", device_Count);
	}
	int device, driver_Version = 0, runtime_Version = 0;
	for (device = 0; device < device_Count; ++device)
	{
		hipSetDevice(device);
		hipDeviceProp_t device_Property;
		hipGetDeviceProperties(&device_Property, device);
		printf("\nDevice %d: \"%s\"\n", device, device_Property.name);
		// Console log
		hipDriverGetVersion(&driver_Version);
		hipRuntimeGetVersion(&runtime_Version);
		printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driver_Version / 1000, (driver_Version % 100) / 10, runtime_Version / 1000, (runtime_Version % 100) / 10);
		printf("  CUDA Capability Major/Minor version number:    %d.%d\n", device_Property.major, device_Property.minor);
		printf("  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
			(float)device_Property.totalGlobalMem / 1048576.0f, (unsigned long long) device_Property.totalGlobalMem);
		printf("  (%2d) Multiprocessors", device_Property.multiProcessorCount);
		printf("  GPU Max Clock rate:      %.0f MHz (%0.2f GHz)\n", device_Property.clockRate * 1e-3f, device_Property.clockRate * 1e-6f);
		// This is supported in CUDA 5.0 (runtime API device properties)
		printf("  Memory Clock rate:                             %.0f Mhz\n", device_Property.memoryClockRate * 1e-3f);
		printf("  Memory Bus Width:                              %d-bit\n", device_Property.memoryBusWidth);
		if (device_Property.l2CacheSize)
		{
			printf("  L2 Cache Size:                                 %d bytes\n", device_Property.l2CacheSize);
		}
		printf("  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n",
			device_Property.maxTexture1D, device_Property.maxTexture2D[0], device_Property.maxTexture2D[1],
			device_Property.maxTexture3D[0], device_Property.maxTexture3D[1], device_Property.maxTexture3D[2]);
		printf("  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
			device_Property.maxTexture1DLayered[0], device_Property.maxTexture1DLayered[1]);
		printf("  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d layers\n",
			   device_Property.maxTexture2DLayered[0], device_Property.maxTexture2DLayered[1], device_Property.maxTexture2DLayered[2]);
		printf("  Total amount of constant memory:               %lu bytes\n", device_Property.totalConstMem);
		printf("  Total amount of shared memory per block:       %lu bytes\n", device_Property.sharedMemPerBlock);
		printf("  Total number of registers available per block: %d\n", device_Property.regsPerBlock);
		printf("  Warp size:                                     %d\n", device_Property.warpSize);
		printf("  Maximum number of threads per multiprocessor:  %d\n", device_Property.maxThreadsPerMultiProcessor);
		printf("  Maximum number of threads per block:           %d\n", device_Property.maxThreadsPerBlock);
		printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
			device_Property.maxThreadsDim[0],
			device_Property.maxThreadsDim[1],
			device_Property.maxThreadsDim[2]);
		printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
			device_Property.maxGridSize[0],
			device_Property.maxGridSize[1],
			device_Property.maxGridSize[2]);
		printf("  Maximum memory pitch:                          %lu bytes\n", device_Property.memPitch);
		printf("  Texture alignment:                             %lu bytes\n", device_Property.textureAlignment);
		printf("  Concurrent copy and kernel execution:          %s with %d copy engine(s)\n", (device_Property.deviceOverlap ? "Yes" : "No"), device_Property.asyncEngineCount);
		printf("  Run time limit on kernels:                     %s\n", device_Property.kernelExecTimeoutEnabled ? "Yes" : "No");
		printf("  Integrated GPU sharing Host Memory:            %s\n", device_Property.integrated ? "Yes" : "No");
		printf("  Support host page-locked memory mapping:       %s\n", device_Property.canMapHostMemory ? "Yes" : "No");
		printf("  Alignment requirement for Surfaces:            %s\n", device_Property.surfaceAlignment ? "Yes" : "No");
		printf("  Device has ECC support:                        %s\n", device_Property.ECCEnabled ? "Enabled" : "Disabled");
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
		printf("  CUDA Device Driver Mode (TCC or WDDM):         %s\n", device_Property.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
#endif
		printf("  Device supports Unified Addressing (UVA):      %s\n", device_Property.unifiedAddressing ? "Yes" : "No");
		printf("  Supports Cooperative Kernel Launch:            %s\n", device_Property.cooperativeLaunch ? "Yes" : "No");
		printf("  Supports MultiDevice Co-op Kernel Launch:      %s\n", device_Property.cooperativeMultiDeviceLaunch ? "Yes" : "No");
		printf("  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n", device_Property.pciDomainID, device_Property.pciBusID, device_Property.pciDeviceID);
		const char *sComputeMode[] =
		{
			"Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
			"Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
			"Prohibited (no host thread can use ::hipSetDevice() with this device)",
			"Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
			"Unknown",
			NULL
		};
		printf("  Compute Mode:\n");
		printf("     < %s >\n", sComputeMode[device_Property.computeMode]);
	}
}
