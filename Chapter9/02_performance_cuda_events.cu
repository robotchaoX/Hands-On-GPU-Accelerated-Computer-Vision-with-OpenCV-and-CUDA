#include "hip/hip_runtime.h"
#include "stdio.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//Defining number of elements in Array
#define N	50000

//Defining Kernel function for vector addition
__global__ void gpuAdd(int *d_a, int *d_b, int *d_c) {
	//Getting Thread index of current kernel
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N)
	{
		d_c[tid] = d_a[tid] + d_b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

int main(void)
{
	//Defining host arrays
	int h_a[N], h_b[N], h_c[N];
	//Defining device pointers
	int *d_a, *d_b, *d_c;
	hipEvent_t e_start, e_stop;
	hipEventCreate(&e_start);
	hipEventCreate(&e_stop);
	hipEventRecord(e_start, 0);
	// allocate the memory
	hipMalloc((void**)&d_a, N * sizeof(int));
	hipMalloc((void**)&d_b, N * sizeof(int));
	hipMalloc((void**)&d_c, N * sizeof(int));
	//Initializing Arrays
	for (int i = 0; i < N; i++)
	{
		h_a[i] = 2 * i*i;
		h_b[i] = i;
	}
	//Copy input arrays from host to device memory
	hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);
	//Calling kernels passing device pointers as parameters
	gpuAdd << <1024, 1024 >> >(d_a, d_b, d_c);
	//Copy result back to host memory from device memory
	hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipEventRecord(e_stop, 0);
	hipEventSynchronize(e_stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, e_start, e_stop);
	printf("Time to add %d numbers: %3.1f ms\n",N, elapsedTime);
	int Correct = 1;
	printf("Vector addition on GPU \n");
	//Printing result on console
	for (int i = 0; i < N; i++)
	{
		if ((h_a[i] + h_b[i] != h_c[i]))
		{
			Correct = 0;
		}
	}
	if (Correct == 1)
	{
		printf("GPU has computed Sum Correctly\n");
	}
	else
	{
		printf("There is an Error in GPU Computation\n");
	}
	//Free up memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
