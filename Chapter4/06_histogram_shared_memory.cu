#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

//Defining number of elements in Array
#define SIZE 1000
// 元素分布范围，每个元素值0-255
#define NUM_BIN 256

__global__ void histogram_shared_memory(int *d_b, int *d_a)
{
	// 全局ID
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	// 3 block * 256 thread=768 thread
	int offset = blockDim.x * gridDim.x;
	// 共享内存
	__shared__ int cache[256];
	// 初始化
	cache[threadIdx.x] = 0;
	__syncthreads();
	while (tid < SIZE)
	{
		// 计数+1
		atomicAdd(&(cache[d_a[tid]]), 1);
		// 一个块内所有线程计算了一次，偏移到下一个片段
		tid += offset;
	}
	__syncthreads();
	// 每个块内的部分统计结果叠加到全局内存上的最终结果
	atomicAdd(&(d_b[threadIdx.x]), cache[threadIdx.x]);
}

int main()
{
	// generate the input array on the host
	int h_a[SIZE];
	for (int i = 0; i < SIZE; i++) {
		//h_a[i] = bit_reverse(i, log2(SIZE));
		h_a[i] = i % NUM_BIN;
	}
	// 定义并初始化结果数组
	int h_b[NUM_BIN];
	for (int i = 0; i < NUM_BIN; i++) {
		h_b[i] = 0;
	}
	// declare GPU memory pointers
	int * d_a;
	int * d_b;
	// allocate GPU memory
	hipMalloc((void **)&d_a, SIZE * sizeof(int));
	hipMalloc((void **)&d_b, NUM_BIN * sizeof(int));
	// transfer the arrays to the GPU
	hipMemcpy(d_a, h_a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, NUM_BIN * sizeof(int), hipMemcpyHostToDevice);
	// launch the kernel
	histogram_shared_memory << <SIZE / 256, 256 >> >(d_b, d_a); // <3,256>
	// copy back the result from GPU
	hipMemcpy(h_b, d_b, NUM_BIN * sizeof(int), hipMemcpyDeviceToHost);
	printf("Histogram using 16 bin is: ");
		for (int i = 0; i < NUM_BIN; i++) {
			printf("bin %d: count %d\n", i, h_b[i]);
		}
	// free GPU memory allocation
	hipFree(d_a);
	hipFree(d_b);
	return 0;
}
